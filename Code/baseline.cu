#include <hip/hip_runtime.h>

__device__ uint32_t Generate(volatile uint64_t &state, uint64_t inc){
    
    // Calculate output function (XSH RR), uses old state for max ILP
    uint32_t xorshifted = ((state >> 18u)^state) >> 27u;
    uint32_t rot = state >> 59u;
    // Update state
    state = (state * 6364136223846793005ULL + 2*inc+1);
    return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}


