#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "baseline.cu"

#define NOUT_PER_THREADS 1

__global__ void Test(uint64_t *DStates, uint32_t *DOuts){

    extern __shared__ uint64_t BStates[];

    /// Read states to shared mem.
    BStates[threadIdx.x] = DStates[blockIdx.x*blockDim.x + threadIdx.x];
    __syncthreads();
    
    #pragma unroll 
    for(int i=0;i<NOUT_PER_THREADS;i++){
        DOuts[(blockIdx.x*blockDim.x + threadIdx.x)*NOUT_PER_THREADS+i] = pcg32_64(BStates[threadIdx.x],blockIdx.x*blockDim.x + threadIdx.x);
    }
    
    
    __syncthreads();
    ///Save states back:
    DStates[blockIdx.x*blockDim.x + threadIdx.x] = BStates[threadIdx.x];


};



int main(int argc, char *argv[]){


    unsigned int BlockSize_x = 256;
    unsigned int GridSize_x = 256;

    /// each thread will have one state.
    uint64_t* HStates = (uint64_t*)malloc(sizeof(uint64_t)*BlockSize_x*GridSize_x);
    uint64_t* DStates;

    uint32_t* HOuts = (uint32_t*)malloc(sizeof(uint32_t)*BlockSize_x*GridSize_x*NOUT_PER_THREADS);
    uint32_t* DOuts;

    /// Allocate device mem.
    if(hipMalloc((void**)&DStates, sizeof(uint64_t)*BlockSize_x*GridSize_x)){
        fprintf(stderr,"ERROR, couldn't allocate Device Mem.%s","\n");
        exit(1);
    }
    if(hipMalloc((void**)&DOuts, sizeof(uint32_t)*BlockSize_x*GridSize_x*NOUT_PER_THREADS)){
        fprintf(stderr,"ERROR, couldn't allocate Device Mem.%s","\n");
        exit(1);
    }


    ///Initialize, all the threads use same seed, the streams are attached to unique tids.
    ///Maximum total threads that can use same seed is limited with 2^63 for 64-bits states.
    ///Each stream geneartes unique RNGS with period 2^64
    uint64_t seed = 99;
    for(unsigned int i=0;i<BlockSize_x*GridSize_x;i++)
        HStates[i] = seed;


    ///Move State -> Dev.
    hipMemcpy(DStates,HStates,sizeof(uint64_t)*BlockSize_x*GridSize_x,hipMemcpyHostToDevice);

    
    ///Launch Kernel:
    for(unsigned int i=0;i<10000;i++)
        Test<<<GridSize_x,BlockSize_x,sizeof(uint64_t)*BlockSize_x>>>(DStates,DOuts);

    printf("Done.%s","\n");

    ///Get Result -> Loc.
    hipMemcpy(HOuts,DOuts,sizeof(uint32_t)*BlockSize_x*GridSize_x*NOUT_PER_THREADS,hipMemcpyDeviceToHost);
    

    free(HStates);
    free(HOuts);
    hipFree(DStates);
    hipFree(DOuts);    
    



    






}





